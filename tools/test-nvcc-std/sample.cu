// sample.cu
//
// This file is part of SHELLKITS.
//
// Copyright (c) 2025 The SHELLKITS project authors. All Rights Reserved.
//
//

#include <hip/hip_runtime.h>
#include <iostream>

__global__ void hello_from_gpu()
{
    printf("test-nvcc!");
}

int main()
{
    hello_from_gpu<<<1, 1>>>();
    hipDeviceSynchronize();
    return 0;
}
